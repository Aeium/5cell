#include "hip/hip_runtime.h"
#include <stdio.h>
#include "lmdb.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>

//#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include <iostream>
//#include "Utilities.cuh"


//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

#define Blocks 640 * 2
#define size 256
#define N Blocks * size

#define CHECK_BIT(var,pos) (var>>pos & 1)

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//
__global__
void autoStep1(unsigned char *a, unsigned char *b, int *direction, unsigned int ruleBase) {

    unsigned int blockBase = blockIdx.x  * size;

    unsigned int tid = blockBase + (threadIdx.x*(size/32));
	
    // rule per block
    // 256 threads per block so 256 cells in automation 

    unsigned int rule = blockIdx.x;
	unsigned int j = threadIdx.x*(size/32);
	
    rule = rule + ruleBase;
	
	for (int i = 0; i<size/32; i++){
	

		unsigned int ll = a[(j+i - 2)% size + blockBase];
		unsigned int l  = a[(j+i - 1)% size + blockBase];
		//unsigned int c  = a[(j+i    )% size + blockBase];
		unsigned int r  = a[(j+i + 1)% size + blockBase];
		unsigned int rr = a[(j+i + 2)% size + blockBase];
		
		
		unsigned  int neighboorhood = (ll << 3) + (l << 2) + (r << 1) + rr;
 
		//if (a[j % blockDim.x + blockBase] == a[tid]){ printf(" MATCH ");}
 
		b[tid+i] = CHECK_BIT(rule, neighboorhood);
		
	    //if(blockIdx.x == 0){
	    //printf("cell: %3d , neigh: %3d , write: %3d\n", i + tid, neighboorhood, b[tid+i]);
		
		//}

		/* the original way was b[tid] == l || b[tid] == ll maybe for some weird reason that's better */
	
		if (b[tid+i] == ll){
				direction[tid+i] -= 1;}
		if (b[tid+i] == l){
				direction[tid+i] -= 1;}
		if (b[tid+i] == rr){
				direction[tid+i] += 1;}
		if (b[tid+i] == r){
				direction[tid+i] += 1;}
	}

}

__global__
void autoStep2(unsigned char *a, unsigned char *b, int *direction, unsigned int ruleBase) {

    unsigned int blockBase = blockIdx.x  * size;

    unsigned int tid = blockBase + (threadIdx.x*(size/32));
	
    // rule per block
    // 256 threads per block so 256 cells in automation 

    unsigned int rule = blockIdx.x;
	unsigned int j = threadIdx.x*size/32;
	
    rule = rule + ruleBase;
	
	for (int i = 0; i<(size/32); i++){
	
		unsigned int ll = b[(j+i - 2)% size + blockBase];
		unsigned int l  = b[(j+i - 1)% size + blockBase];
		//unsigned int c  = b[(j+i    )% size + blockBase];
		unsigned int r  = b[(j+i + 1)% size + blockBase];
		unsigned int rr = b[(j+i + 2)% size + blockBase];
  
		unsigned  int neighboorhood = (ll << 3) + (l << 2) + (r << 1) + rr;
 
		//if (a[j % blockDim.x + blockBase] == a[tid]){ printf(" MATCH ");}
 
		a[tid+i] = CHECK_BIT(rule, neighboorhood);
		
	    //if(blockIdx.x == 0){
	    //printf("cell: %3d , neigh: %3d , write: %3d\n", i + tid, neighboorhood, b[tid+i]);
		
		//}
		

		/* the original way was b[tid] == l || b[tid] == ll maybe for some weird reason that's better */
	
		if (a[tid+i] == ll){
				direction[tid+i] -= 1;}
		if (a[tid+i] == l){
				direction[tid+i] -= 1;}
		if (a[tid+i] == rr){
				direction[tid+i] += 1;}
		if (a[tid+i] == r){
				direction[tid+i] += 1;}
	}

}


//30

// 128  64  32  16   8   4   2   1
//  0   0    0   1   1   1   1   0
 
 
// 1024, 1024, 64
 
int main( int argc, char *argv[] ) {
    //
    // Create int arrays on the CPU.
    // ('h' stands for "host".)
    //
    unsigned char ha[N], hb[N];

    int directionArray[N];
	
	// DB allocations
	/*
	int rc;
	MDB_env *env;
	MDB_dbi dbi;
	MDB_val key, data;
	MDB_txn *txn;
	MDB_cursor *cursor;
	
	rc = mdb_env_create(&env);
	rc = mdb_env_set_mapsize(env, (68719476736 * 2));
	rc = mdb_env_open(env, "./automationDB", 0, 0664);
	*/
	
    //
    // Create corresponding int arrays on the GPU.
    // ('d' stands for "device".)
    //
    unsigned char *da, *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(char));

    int *directionArrayD;
    hipMalloc((void **)&directionArrayD, N*sizeof(int));

    //
    // Initialise the input data on the CPU.
    //
	
    srand(time(NULL) + 5);
	
    for (int i = 0; i<N; ++i) {
        ha[i] = rand()%2;
		//ha[i] = 0;
        //if(i%256 - 128 == 0){ ha[i] = 1;}
		
    }
    for (int i = 0; i<N; ++i) {
        directionArray[i] = 0;
    }
    for (int i = 0; i<N; ++i) {
        hb[i] = 0;
    }
	
	//ha[] = 1;
    //ha[] = 1;
	
    //
    // Copy input data to array on GPU.
    //
    hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(directionArrayD, directionArray, N*sizeof(int), hipMemcpyHostToDevice);
    //
    // Launch GPU code with N threads, one per
    // array element.
    //
	
    //for (int i = 0; i<N; ++i) {
    //    printf("%d", ha[i]);
    //}
	
	// 
	
    unsigned int rule = atoi(argv[1]);  //2769352740;//2769351763;  // + 465 is best in local range
    //                  2769351843
	
	printf("\n returning values:\n");
	
	int scan = 0; //atoi(argv[1]) *size;
	
	int sum = 0;
		
	    for (int i = 0+scan; i<size+scan; ++i) {
			if(ha[i] == 1){ printf("X");}
			else          { printf(" ");}
        }
		
		//printf("\n");
		
	    for (int i = 0+scan; i<size+scan; ++i) {
	        //printf ("%3d  ", directionArray[i]);
            sum += directionArray[i];
        }
		
	    printf("sum: %d\n", sum);
	
	
	for (int i = 0; i<40; i++) {


		autoStep1<<<Blocks, 32>>>(da, db, directionArrayD, rule);
		
	    hipMemcpy(hb, db, N*sizeof(char), hipMemcpyDeviceToHost);
	    hipMemcpy(directionArray, directionArrayD, N*sizeof(int), hipMemcpyDeviceToHost);
	
		sum = 0;
		
	    for (int i = 0+scan; i<size+scan; ++i) {
			if(hb[i] == 1){ printf("X");}
			else          { printf(" ");}
        }
		
		//printf("\n");
	
	    for (int i = 0+scan; i<size+scan; ++i) {
	        //printf ("%3d  ", directionArray[i]);
            sum += directionArray[i];
        }
	
	    printf("sum: %d\n", sum);
		
		
		autoStep2<<<640, 32>>>(da, db, directionArrayD, rule);
	
	    hipMemcpy(ha, da, N*sizeof(char), hipMemcpyDeviceToHost);
	    hipMemcpy(directionArray, directionArrayD, N*sizeof(int), hipMemcpyDeviceToHost);
		
		sum = 0;
		
	    for (int i = 0+scan; i<size+scan; ++i) {
			if(ha[i] == 1){ printf("X");}
			else          { printf(" ");}
        }
		
		//printf("\n");
		
	    for (int i = 0+scan; i<size+scan; ++i) {
	        //printf ("%3d  ", directionArray[i]);
            sum += directionArray[i];
        }
		
	    printf("sum: %d\n", sum);
	

	
	}
	
	hipMemcpy(ha, da, N*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(directionArray, directionArrayD, N*sizeof(int), hipMemcpyDeviceToHost);
	
    //for (int i = 0; i<100; ++i) {
	//	printf("%d", ha[i]);
	//}

		
	printf("\nDirection: ");
	
    sum = 0;
    sum = 0;
	
	for (int i = 0+scan; i<size+scan; ++i) {
	    //printf (" %d ", directionArray[i]);
        sum += directionArray[i];
    }
	
	printf("Direction Sum: %d\n", sum);

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}

// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100
// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100