#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "lmdb.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <chrono>
#include <cudaProfiler.h>
#include <lmdb.h>



using Clock = std::chrono::steady_clock;
using std::chrono::time_point;
using std::chrono::duration_cast;
using std::chrono::milliseconds;


//#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include <iostream>
//#include "Utilities.cuh"


//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

#define N 20480
#define size 256
#define CHECK_BIT(var,pos) (var>>pos & 1)

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//
__global__
void autoStep1(int *a, int *b, int *direction, unsigned int ruleBase) {

    unsigned int blockBase = blockIdx.x  * blockDim.x;

    unsigned int tid = blockBase + threadIdx.x;
	
    // rule per block
    // 256 threads per block so 256 cells in automation 

    unsigned int rule = blockIdx.x;
	unsigned int j = threadIdx.x;
	
    rule = rule + ruleBase;
	
    unsigned int ll = a[(j - 2)% blockDim.x + blockBase];
    unsigned int l  = a[(j - 1)% blockDim.x + blockBase];
    unsigned int c  = a[(j    )% blockDim.x + blockBase];
    unsigned int r  = a[(j + 1)% blockDim.x + blockBase];
    unsigned int rr = a[(j + 2)% blockDim.x + blockBase];
  
    unsigned  int neighboorhood = ll * 16 + l * 8 + c * 4 + r * 2 + rr;
 
    //if (a[j % blockDim.x + blockBase] == a[tid]){ printf(" MATCH ");}
 
    b[tid] = CHECK_BIT(rule, neighboorhood);

    /* the original way was b[tid] == l || b[tid] == ll maybe for some weird reason that's better */
	
    if (b[tid] == ll){
			direction[tid] -= 1;}
    if (b[tid] == l){
			direction[tid] -= 1;}
    if (b[tid] == rr){
			direction[tid] += 1;}
    if (b[tid] == r){
			direction[tid] += 1;}

}

__global__
void autoStep2(int *a, int *b, int *direction, unsigned int ruleBase) {

    unsigned int blockBase = blockIdx.x  * blockDim.x;

    unsigned int tid = blockBase + threadIdx.x;
	
    // rule per block
    // 256 threads per block so 256 cells in automation 

    unsigned int rule = blockIdx.x;
	unsigned int j = threadIdx.x;
	
    rule = rule + ruleBase;
	
    unsigned int ll = b[(j - 2)% blockDim.x + blockBase];
    unsigned int l  = b[(j - 1)% blockDim.x + blockBase];
    unsigned int c  = b[(j    )% blockDim.x + blockBase];
    unsigned int r  = b[(j + 1)% blockDim.x + blockBase];
    unsigned int rr = b[(j + 2)% blockDim.x + blockBase];
  
    unsigned  int neighboorhood = ll * 16 + l * 8 + c * 4 + r * 2 + rr;
 
    a[tid] = CHECK_BIT(rule, neighboorhood);
	
    if (a[tid] == ll){
			direction[tid] -= 1;}
    if (a[tid] == l){
			direction[tid] -= 1;}
    if (a[tid] == rr){
			direction[tid] += 1;}
    if (a[tid] == r){
			direction[tid] += 1;}
}


//30

// 128  64  32  16   8   4   2   1
//  0   0    0   1   1   1   1   0
 
 
// 1024, 1024, 64
 
int main( int argc, char *argv[] ) {

    //
    // Create int arrays on the CPU.
    // ('h' stands for "host".)
    //
    int ha[N], hb[N];

    int directionArray[N];
	
    //
    // Create corresponding int arrays on the GPU.
    // ('d' stands for "device".)
    //
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    int *directionArrayD;
    hipMalloc((void **)&directionArrayD, N*sizeof(int));

    time_point<Clock> start = Clock::now();
	
    //
    // Initialise the input data on the CPU.
    //
	
	//srand(time(NULL));

	int score1[80];
	int score2[80];
	
	int finalScore[80];
	
	unsigned int rule = 2769351763;

    int loop = 1;
	
	while (loop){
	
		for (int i = 0; i<80; ++i) {
			score1[i] = 0;
			score2[i] = 0;
		}
	
		for (int iters = 0; iters<5000; iters++) {
	
			srand(time(NULL) + iters);
	
			for (int i = 0; i<N; ++i) {
				ha[i] = rand()%2;
				//if(i%256 - 128 == 0){ ha[i] = 1;}
			}

			for (int i = 0; i<N; ++i) {
				hb[i] = 0;
			}
	
			for (int i = 0; i<N; ++i) {
				directionArray[i] = 0;
			}

			//
			// Copy input data to array on GPU.
			//
			hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(directionArrayD, directionArray, N*sizeof(int), hipMemcpyHostToDevice);
			//
			// Launch GPU code with N threads, one per
			// array element.
			//
	
			int scan = 0;
	
			unsigned int sum = 0;
	
			for (int i = 0; i<40; i++) {

				autoStep1<<<80, 256>>>(da, db, directionArrayD, rule);
				autoStep2<<<80, 256>>>(da, db, directionArrayD, rule);

			}
	
			hipMemcpy(ha, da, N*sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(directionArray, directionArrayD, N*sizeof(int), hipMemcpyDeviceToHost);
	
			sum = 0;
	
			int count = 0;
	
			for (int i = 0+scan; i<N; ++i) {
				//printf (" %d ", directionArray[i]);
				if(i%256 == 0){
		       
					score1[count] += sum;
					score2[count] += abs(int(sum));
				
					sum = 0;
					count++;
				}
				sum += directionArray[i];
			}
		
		}

		/*
		for (int i = 0; i<80; i++){
	
			printf("score1: %8d, score2: %8d\n", score1[i], score2[i]);
	
			finalScore[i] = score2[i] - abs(score1[i]);
		
			if (finalScore[i] > 0) { loop = 0;}
		}
		*/
	
		std::cout << "before db" << std::endl;
	
		// write results to db here
		char keyValue[16];
		char dataValue[16];
	
		std::cout << "db 1" << std::endl;
	
		rc = mdb_txn_begin(env, NULL, 0, &txn);
		rc = mdb_open(txn, NULL, 0, &dbi);
	
		std::cout << "db 2" << std::endl;
	
		key.mv_size = 8;
		key.mv_data = keyValue;
		data.mv_size = 16;
		data.mv_data = dataValue;
	
		unsigned int autoID;
	
		std::cout << "db 3" << std::endl;
	
		for(int i = 0; i < 80; i++){
	
			//std::cout << "1 writing to db #" << i << std::endl;
	
			sprintf(keyValue,  "%08x", results[i].autoID);
			sprintf(dataValue,  "%08.0f", results[i].score);
		
			//std::cout << "Key: " << keyValue << " Score: " << dataValue << std::endl;
	
			//std::cout << "2 writing to db #" << i << std::endl;
	
			rc = mdb_put(txn, dbi, &key, &data, 0);
	
		}
		
		rc = mdb_txn_commit(txn);
	
		if (rc) {
			fprintf(stderr, "mdb_txn_commit: (%d) %s\n", rc, mdb_strerror(rc));
			goto leave;
		}
	
		printf("MDB commit rc#: %d\n", rc);
	
		std::cout << "db 4" << std::endl;
	
		rule += 80;
	
	}
	
    time_point<Clock> end = Clock::now();
	
	for (int i = 0; i<80; i++){
	
		printf("auto %u, score: %d\n", rule + i, finalScore[i]);
	}
	
	//printf("Direction Sum: %d\n", sum);
	
    milliseconds diff = duration_cast<milliseconds>(end - start);
    std::cout << diff.count() << "ms" << std::endl;

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}

// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100
// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100