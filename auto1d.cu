#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "lmdb.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <chrono>
#include <cudaProfiler.h>
#include <lmdb.h>
#include <fstream>



using Clock = std::chrono::steady_clock;
using std::chrono::time_point;
using std::chrono::duration_cast;
using std::chrono::milliseconds;


//#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include <iostream>
//#include "Utilities.cuh"


//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

#define Blocks 640 * 2
#define size 128
#define N Blocks * size
#define CHECK_BIT(var,pos) (var>>pos & 1)
#define time 
#define PLUS_ONE 1.0f
#define MINUS_ONE -1.0f

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//
__global__
void autoStep1(unsigned char *a, unsigned char *b, int *direction, unsigned int ruleBase) {

    unsigned int blockBase = blockIdx.x  * 128;

    unsigned int tid = blockBase + threadIdx.x; //(threadIdx.x*8);
	
    // rule per block
    // 256 threads per block so 256 cells in automation 
	
	
	unsigned char neighboorhood = ((a[(threadIdx.x - 2)% 128 + blockBase]) << 4) +
			                      ((a[(threadIdx.x - 1)% 128 + blockBase]) << 3) +
	    	                      ((a[(threadIdx.x    )% 128 + blockBase]) << 2) +
			                      ((a[(threadIdx.x + 1)% 128 + blockBase]) << 1) +
                                    a[(threadIdx.x + 2)% 128 + blockBase];
    
	
	b[tid] = CHECK_BIT(blockIdx.x + ruleBase, neighboorhood );
	
    //half one = __float2half(ONE);
	
	if (b[tid] == CHECK_BIT( neighboorhood , 4 )){
			direction[tid] -= 1;}
	if (b[tid] == CHECK_BIT( neighboorhood , 3 )){
			direction[tid] -= 1;}
	if (b[tid] == CHECK_BIT( neighboorhood , 1 )){
			direction[tid] += 1;}
	if (b[tid] == CHECK_BIT( neighboorhood , 0 )){
			direction[tid] += 1;}
			

	/*
	unsigned char ll = a[(threadIdx.x - 2)% 128 + blockBase];
	unsigned char l  = a[(threadIdx.x - 1)% 128 + blockBase];
	unsigned char c  = a[(threadIdx.x    )% 128 + blockBase];
	unsigned char r  = a[(threadIdx.x + 1)% 128 + blockBase];
	unsigned char rr = a[(threadIdx.x + 2)% 128 + blockBase];
  
	b[tid] = (ll << 4) + (l << 3) + (c << 2) + (r << 1) + (rr);//ll * 16 + l * 8 + c * 4 + r * 2 + rr;
 
	b[tid] = CHECK_BIT(blockIdx.x + ruleBase, b[tid]);

		// the original way was b[tid] == l || b[tid] == ll maybe for some weird reason that's better 
	
	
	//if (b[tid] == ll || b[tid] == l){
	//	direction[tid] -= 1;}
	//  if (b[tid] == rr || b[tid] == r){
	//	direction[tid] += 1;}
	
	
	
	if (b[tid] == ll){
			direction[tid] -= 1;}
	if (b[tid] == l){
			direction[tid] -= 1;}
	if (b[tid] == rr){
			direction[tid] += 1;}
	if (b[tid] == r){
			direction[tid] += 1;}
			
    */
			
}

__global__
void autoStep2(unsigned char *a, unsigned char *b, int *direction, unsigned int ruleBase) {

    unsigned int blockBase = blockIdx.x  * 128;

    unsigned int tid = blockBase + threadIdx.x;
	
    // rule per block
    // 256 threads per block so 256 cells in automation 
	
	
	unsigned char neighboorhood = ((b[(threadIdx.x - 2)% 128 + blockBase]) << 4) +
			                      ((b[(threadIdx.x - 1)% 128 + blockBase]) << 3) +
	    	                      ((b[(threadIdx.x    )% 128 + blockBase]) << 2) +
			                      ((b[(threadIdx.x + 1)% 128 + blockBase]) << 1) +
                                   b[(threadIdx.x + 2)% 128 + blockBase];
    
	a[tid] = CHECK_BIT(blockIdx.x + ruleBase, neighboorhood );
	
    //half one = __float2half(ONE);
	
	if (a[tid] == CHECK_BIT( neighboorhood , 4 )){
			direction[tid] -= 1;}
	if (a[tid] == CHECK_BIT( neighboorhood , 3 )){
			direction[tid] -= 1;}
	if (a[tid] == CHECK_BIT( neighboorhood , 1 )){
			direction[tid] += 1;}
	if (a[tid] == CHECK_BIT( neighboorhood , 0 )){
			direction[tid] += 1;}
			
			
	/*
	
	
	unsigned char ll = b[(threadIdx.x - 2)% 128 + blockBase];
	unsigned char l  = b[(threadIdx.x - 1)% 128 + blockBase];
	unsigned char c  = b[(threadIdx.x    )% 128 + blockBase];
	unsigned char r  = b[(threadIdx.x + 1)% 128 + blockBase];
	unsigned char rr = b[(threadIdx.x + 2)% 128 + blockBase];
  
	a[tid] = (ll << 4) + (l << 3) + (c << 2) + (r << 1) + (rr);//ll * 16 + l * 8 + c * 4 + r * 2 + rr; //ll << 4 + l << 3 + c << 2 + r << 1 + rr;
 
	a[tid] = CHECK_BIT(blockIdx.x + ruleBase, a[tid]);

		// the original way was b[tid] == l || b[tid] == ll maybe for some weird reason that's better
	
	
	//if (a[tid] == ll || a[tid] == l){
	//	direction[tid] -= 1;}
	//if (a[tid] == rr || a[tid] == r){
	//		direction[tid] += 1;}
	
	
	if (a[tid] == ll){
			direction[tid] -= 1;}
	if (a[tid] == l){
			direction[tid] -= 1;}
	if (a[tid] == rr){
			direction[tid] += 1;}
	if (a[tid] == r){
			direction[tid] += 1;}
    */
 
}


__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    hiprand_init(1231+id, id, 0, &state[id]);
}

__global__ void curand_kernel(hiprandState *state,
                                unsigned char *a)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    float x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    x = (hiprand(&localState))%2;
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store last generated result per thread */
    a[id] = (unsigned char) x;
}


//30

// 128  64  32  16   8   4   2   1
//  0   0    0   1   1   1   1   0
 
 
// 1024, 1024, 64
 
int main( int argc, char *argv[] ) {


    // DB allocations

	int rc;
	MDB_env *env;
	MDB_dbi dbi;
	MDB_val key, data;
	MDB_txn *txn;
	MDB_cursor *cursor;
	
	rc = mdb_env_create(&env);
	rc = mdb_env_set_mapsize(env, (68719476736 * 2));
	rc = mdb_env_open(env, argv[2], 0, 0664);


    //
    // Create int arrays on the CPU.
    // ('h' stands for "host".)
    //
    unsigned char ha[N];

    int directionArray[N];
	
    //
    // Create corresponding int arrays on the GPU.
    // ('d' stands for "device".)
    //
    hiprandState *devStates;
    hipMalloc((void **)&devStates, N *  sizeof(hiprandState));
	
    setup_kernel<<<Blocks, size>>>(devStates);
	
	unsigned char *da, *db;
    hipMalloc((void **)&da, N*sizeof(char));
    hipMalloc((void **)&db, N*sizeof(char));

	
	
    int *directionArrayD;
    hipMalloc((void **)&directionArrayD, N*sizeof(int));

    //time_point<Clock> start = Clock::now();
	
    //
    // Initialise the input data on the CPU.
    //
	
	//srand(time(NULL));

	int score1[Blocks];
	int score2[Blocks];
	
	int finalScore[Blocks];
	
	unsigned int rule = atoi( argv[1]);
	//                  2769351843
	//                  2769352740

    std::ofstream logs;
	
    int loop = 0;
	
	while (loop < 1677722){
	
		for (int i = 0; i<Blocks; ++i) {
			score1[i] = 0;
			score2[i] = 0;
		}
	
		for (int iters = 0; iters<5; iters++) {
	
			srand(time(NULL) + iters);
	
	        // http://stackoverflow.com/questions/14289378/generating-random-numbers-within-cuda-kernel
			
			// GENERATING RANDOM NUMBERS WITH CUDA WILL SPEED THIS UP ALOT MAYBE
	
	
			//for (int i = 0; i<N; ++i) {
			//	ha[i] = rand()%2;
				//if(i%256 - 128 == 0){ ha[i] = 1;}
			//}

			//for (int i = 0; i<N; ++i) {
			//	hb[i] = 0;
			//}
	
			//for (int i = 0; i<N; ++i) {
			//	directionArray[i] = 0;
			//}

			//
			// Copy input data to array on GPU.
			//
			//hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
			
			hipMemset(da, 0, N * sizeof(char));

            curand_kernel<<<Blocks, size>>>(devStates, da);
			
            //hipMemcpy(ha, da, N*sizeof(char), hipMemcpyDeviceToHost);		
			
			hipMemset(directionArrayD, 0, N*sizeof(int));
			//hipMemcpy(directionArrayD, directionArray, N*sizeof(char), hipMemcpyHostToDevice);
			//
			// Launch GPU code with N threads, one per
			// array element.
			//
	
			int scan = 0;
	
			int sum = 0;
	
			for (int i = 0; i<40; i++) {

				autoStep1<<<Blocks, size>>>(da, db, directionArrayD, rule);
				autoStep2<<<Blocks, size>>>(da, db, directionArrayD, rule);

			}
	
			//hipMemcpy(ha, da, N*sizeof(char), hipMemcpyDeviceToHost);
			hipMemcpy(directionArray, directionArrayD, N*sizeof(int), hipMemcpyDeviceToHost);
	
			sum = 0;
	
			int count = 0;
	
			for (int i = 0+scan; i<N; ++i) {
				if(i%size == 0 && i > 0){
		       
					score1[count] += sum;
					score2[count] += abs(int(sum));
				
                    //printf("auto %ud score2: %d ", rule + count, score2[count]);
				
					sum = 0;
					count++;
				}
				sum += directionArray[i];
			}
			
			//count++;
			
			score1[count] += sum;
			score2[count] += abs(int(sum));
				
		
		}

		
		for (int i = 0; i<(N/size); i++){
	
			//printf("score1: %8d, score2: %8d\n", score1[i], score2[i]);
	
			finalScore[i] = score2[i] - abs(score1[i]);
	        //printf(" %d , %u , %d \n" , i , rule + i, finalScore[i]);
		
			//if (finalScore[i] > 0) { loop = 0;}
		}
	
        time_point<Clock> end = Clock::now();
	
		//std::cout << "before db" << std::endl;
	
		// write results to db here
		char keyValue[16];
		char dataValue[16];
	
		//std::cout << "db 1" << std::endl;
	
		rc = mdb_txn_begin(env, NULL, 0, &txn);
		rc = mdb_open(txn, NULL, 0, &dbi);
	
		//std::cout << "db 2" << std::endl;
	
		key.mv_size = 8;
		key.mv_data = keyValue;
		data.mv_size = 8;
		data.mv_data = dataValue;
	
		//unsigned int autoID;
	
		//std::cout << "db 3" << std::endl;
	
		for(int i = 0; i < (Blocks); i++){
	
			//std::cout << "1 writing to db #" << i << std::endl;
	
			sprintf(keyValue,  "%08x", rule + i);
			sprintf(dataValue,  "%08x", finalScore[i]);
		
			//std::cout << "Key: " << keyValue << " Score: " << dataValue << std::endl;
	
			//std::cout << "2 writing to db #" << i << std::endl;
	
			rc = mdb_put(txn, dbi, &key, &data, 0);
	
		}
		
		rc = mdb_txn_commit(txn);
	
		if (rc) {
			fprintf(stderr, "mdb_txn_commit: (%d) %s\n", rc, mdb_strerror(rc));
			mdb_close(env, dbi);
			mdb_env_close(env);
			return 0;
		}
	
		//printf("MDB commit rc#: %d\n", rc);
	
		//std::cout << "db 4" << std::endl;
	   
	
		if ((rule % 124000) == 0) { 
			printf("Completed section from Rule %d, loop %d\n", rule, loop); 
			logs.open("auto1d_log2.txt");
			logs << "Completed section from Rule %d, loop %d\n", rule, loop;
			logs.close();
		}
		loop++;
	
		rule += Blocks;
	
	}
	
	//for (int i = 0; i<N/size; i++){
	
	//	printf("auto %u, score: %d\n", rule + i, finalScore[i]);
	//}
	
	//printf("Direction Sum: %d\n", sum);
	
    //milliseconds diff = duration_cast<milliseconds>(end - start);
    //std::cout << diff.count() << "ms" << std::endl;

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}



// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100
// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100