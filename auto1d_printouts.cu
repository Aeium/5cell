#include "hip/hip_runtime.h"
#include <stdio.h>
#include "lmdb.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>

//#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include <iostream>
//#include "Utilities.cuh"


//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

#define N 163840
#define size 256
#define CHECK_BIT(var,pos) (var>>pos & 1)

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//
__global__
void autoStep1(int *a, int *b, int *direction, unsigned int ruleBase) {

    unsigned int blockBase = blockIdx.x  * 256;

    unsigned int tid = blockBase + (threadIdx.x*8);
	
    // rule per block
    // 256 threads per block so 256 cells in automation 

    unsigned int rule = blockIdx.x;
	unsigned int j = threadIdx.x*8;
	
    rule = rule + ruleBase;
	
	for (int i = 0; i<8; i++){
		unsigned int ll = a[(j+i - 2)% 256 + blockBase];
		unsigned int l  = a[(j+i - 1)% 256 + blockBase];
		unsigned int c  = a[(j+i    )% 256 + blockBase];
		unsigned int r  = a[(j+i + 1)% 256 + blockBase];
		unsigned int rr = a[(j+i + 2)% 256 + blockBase];
  
		unsigned  int neighboorhood = ll * 16 + l * 8 + c * 4 + r * 2 + rr;
 
		//if (a[j % blockDim.x + blockBase] == a[tid]){ printf(" MATCH ");}
 
		b[tid+i] = CHECK_BIT(rule, neighboorhood);
		
	    //if(blockIdx.x == 0){
	    //printf("cell: %3d , neigh: %3d , write: %3d\n", i + tid, neighboorhood, b[tid+i]);
		
		//}

		/* the original way was b[tid] == l || b[tid] == ll maybe for some weird reason that's better */
	
		if (b[tid+i] == ll){
				direction[tid+i] -= 1;}
		if (b[tid+i] == l){
				direction[tid+i] -= 1;}
		if (b[tid+i] == rr){
				direction[tid+i] += 1;}
		if (b[tid+i] == r){
				direction[tid+i] += 1;}
	}

}

__global__
void autoStep2(int *a, int *b, int *direction, unsigned int ruleBase) {

    unsigned int blockBase = blockIdx.x  * 256;

    unsigned int tid = blockBase + (threadIdx.x*8);
	
    // rule per block
    // 256 threads per block so 256 cells in automation 

    unsigned int rule = blockIdx.x;
	unsigned int j = threadIdx.x*8;
	
    rule = rule + ruleBase;
	
	for (int i = 0; i<8; i++){
	
		unsigned int ll = b[(j+i - 2)% 256 + blockBase];
		unsigned int l  = b[(j+i - 1)% 256 + blockBase];
		unsigned int c  = b[(j+i    )% 256 + blockBase];
		unsigned int r  = b[(j+i + 1)% 256 + blockBase];
		unsigned int rr = b[(j+i + 2)% 256 + blockBase];
  
		unsigned  int neighboorhood = ll * 16 + l * 8 + c * 4 + r * 2 + rr;
		
 
		//if (a[j % blockDim.x + blockBase] == a[tid]){ printf(" MATCH ");}
 
		a[tid+i] = CHECK_BIT(rule, neighboorhood);
		
	    //if(blockIdx.x == 0){
	    //printf("cell: %3d , neigh: %3d , write: %3d\n", i + tid, neighboorhood, b[tid+i]);
		
		//}
		

		/* the original way was b[tid] == l || b[tid] == ll maybe for some weird reason that's better */
	
		if (a[tid+i] == ll){
				direction[tid+i] -= 1;}
		if (a[tid+i] == l){
				direction[tid+i] -= 1;}
		if (a[tid+i] == rr){
				direction[tid+i] += 1;}
		if (a[tid+i] == r){
				direction[tid+i] += 1;}
	}

}


//30

// 128  64  32  16   8   4   2   1
//  0   0    0   1   1   1   1   0
 
 
// 1024, 1024, 64
 
int main( int argc, char *argv[] ) {
    //
    // Create int arrays on the CPU.
    // ('h' stands for "host".)
    //
    int ha[N], hb[N];

    int directionArray[N];
	
	// DB allocations
	/*
	int rc;
	MDB_env *env;
	MDB_dbi dbi;
	MDB_val key, data;
	MDB_txn *txn;
	MDB_cursor *cursor;
	
	rc = mdb_env_create(&env);
	rc = mdb_env_set_mapsize(env, (68719476736 * 2));
	rc = mdb_env_open(env, "./automationDB", 0, 0664);
	*/
	
    //
    // Create corresponding int arrays on the GPU.
    // ('d' stands for "device".)
    //
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    int *directionArrayD;
    hipMalloc((void **)&directionArrayD, N*sizeof(int));

    //
    // Initialise the input data on the CPU.
    //
    for (int i = 0; i<N; ++i) {
        ha[i] = 0;
        if(i%256 - 128 == 0){ ha[i] = 1;}
		
    }
    for (int i = 0; i<N; ++i) {
        directionArray[i] = 0;
    }
    for (int i = 0; i<N; ++i) {
        hb[i] = 0;
    }
	
	//ha[] = 1;
    //ha[] = 1;
	
    //
    // Copy input data to array on GPU.
    //
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(directionArrayD, directionArray, N*sizeof(int), hipMemcpyHostToDevice);
    //
    // Launch GPU code with N threads, one per
    // array element.
    //
	
    //for (int i = 0; i<N; ++i) {
    //    printf("%d", ha[i]);
    //}
	
    unsigned int rule = 2769351767;
	
	printf("\n returning values:\n");
	
	int scan = atoi( argv[1]) *size;
	
	unsigned int sum = 0;
	
		sum = 0;
		
	    for (int i = 0+scan; i<256+scan; ++i) {
			if(ha[i] == 1){ printf("X");}
			else          { printf(".");}
        }
		
		//printf("\n");
		
	    for (int i = 0+scan; i<256+scan; ++i) {
	        //printf ("%3d  ", directionArray[i]);
            sum += directionArray[i];
        }
		
	    printf("sum: %d\n", sum);
	
	
	for (int i = 0; i<40; i++) {


		autoStep1<<<640, 32>>>(da, db, directionArrayD, rule);
		
	    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
	    hipMemcpy(directionArray, directionArrayD, N*sizeof(int), hipMemcpyDeviceToHost);
	
		sum = 0;
		
	    for (int i = 0+scan; i<256+scan; ++i) {
			if(hb[i] == 1){ printf("X");}
			else          { printf(" ");}
        }
		
		//printf("\n");
	
	    for (int i = 0+scan; i<256+scan; ++i) {
	        //printf ("%3d  ", directionArray[i]);
            sum += directionArray[i];
        }
	
	    printf("sum: %d\n", sum);
		
		
		autoStep2<<<640, 32>>>(da, db, directionArrayD, rule);
	
	    hipMemcpy(ha, da, N*sizeof(int), hipMemcpyDeviceToHost);
	    hipMemcpy(directionArray, directionArrayD, N*sizeof(int), hipMemcpyDeviceToHost);
		
		sum = 0;
		
	    for (int i = 0+scan; i<256+scan; ++i) {
			if(ha[i] == 1){ printf("X");}
			else          { printf(" ");}
        }
		
		//printf("\n");
		
	    for (int i = 0+scan; i<256+scan; ++i) {
	        //printf ("%3d  ", directionArray[i]);
            sum += directionArray[i];
        }
		
	    printf("sum: %d\n", sum);
	

	
	}
	
	hipMemcpy(ha, da, N*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(directionArray, directionArrayD, N*sizeof(int), hipMemcpyDeviceToHost);
	
    //for (int i = 0; i<100; ++i) {
	//	printf("%d", ha[i]);
	//}

		
	printf("\nDirection: ");
	
    sum = 0;
    sum = 0;
	
	for (int i = 0+scan; i<256+scan; ++i) {
	    //printf (" %d ", directionArray[i]);
        sum += directionArray[i];
    }
	
	printf("Direction Sum: %d\n", sum);

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}

// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100
// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100