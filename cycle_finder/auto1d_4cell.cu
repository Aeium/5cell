#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "lmdb.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <chrono>
#include <cudaProfiler.h>
#include <lmdb.h>
#include <fstream>
#include "addressbook.pb.h"


// ./4cell_debug1 65376 123

// vs 

// ./4cell_debug1 0 123

// should be printing out results of same automation but they aint


using Clock = std::chrono::steady_clock;
using std::chrono::time_point;
using std::chrono::duration_cast;
using std::chrono::milliseconds;


//#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include <iostream>
//#include "Utilities.cuh"


//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

#define blockMult 2
#define Blocks (640 * blockMult)
#define size 128
#define N (Blocks * size)
#define CHECK_BIT(var,pos) (var>>pos & 1)
#define time 
#define PLUS_ONE    1.0f
#define MINUS_ONE   -1.0f
#define simDuration 400
#define rounds      100

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//

__global__
void combinedKernel(unsigned char *a1, unsigned char *a2, 
 unsigned char *b1, unsigned char *b2, int *direction, unsigned int ruleBase) {

    unsigned int rule = blockBase + threadIdx.x; //(threadIdx.x*8);
	
    // rule per block
    // 256 threads per block so 256 cells in automation 
	
	unsigned char a1[size];
	unsigned char a2[size];

	unsigned char b1[size];
	unsigned char b2[size];
	
	int index1;
	int index2;
	int index3;
	int index4;
	
	unsigned char mask;
	
	    // im going to pack 8 bits in each char to save some memory	
	
        // it's a bit hard to visualize because the bit order is opposite between bits in chars and chars, but I don't think it actually matters
	
	for (int j = 0; j < rounds; j ++){
	
	    // TURTLE STEP 1
	
	    for (int i = 0; i < size * 8; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhoodT = CHECK_BIT( a1[(i-2)/8], (i-2)%8) << 3 +    
		           	                       CHECK_BIT( a1[(i-1)/8], (i-1)%8) << 2 +
		    	                           CHECK_BIT( a1[(i+1)/8], (i+1)%8) << 1 +
                                           CHECK_BIT( a1[(i+2)/8], (i+2)%8);
	
	        // 1111 1110 is 254
		
	    	mask = 254;                                                             // defalt mask for no bit shift
	
	        mask = (mask >> 8-i%8) | (mask << i%8);                                 // create proper mask to preserve other bits
	
            a2[i/8] = (a2[i/8] & mask) + ( CHECK_BIT(rule, neighboorhood) << i%8);  // use mask and add in new bit
	
	    }
	
	    // TURTLE STEP 2
	
	    for (int i = 0; i < size * 8; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhoodT = CHECK_BIT( a2[(i-2)/8], (i-2)%8) << 3 +    
		        	                       CHECK_BIT( a2[(i-1)/8], (i-1)%8) << 2 +
		    	                           CHECK_BIT( a2[(i+1)/8], (i+1)%8) << 1 +
                                          CHECK_BIT( a2[(i+2)/8], (i+2)%8);
	
	        // 1111 1110 is 254
		
		    mask = 254;                                                             // defalt mask for no bit shift
	
	        mask = (mask >> 8-i%8) | (mask << i%8);                                 // create proper mask to preserve other bits
	
            a1[i/8] = (a1[i/8] & mask) + ( CHECK_BIT(rule, neighboorhood) << i%8);  // use mask and add in new bit
	
	    }
	
	    // RABBIT STEP 1
	
	    for (int i = 0; i < size * 8; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhoodT = CHECK_BIT( b1[(i-2)/8], (i-2)%8) << 3 +    
		         	                       CHECK_BIT( b1[(i-1)/8], (i-1)%8) << 2 +
		    	                           CHECK_BIT( b1[(i+1)/8], (i+1)%8) << 1 +
                                           CHECK_BIT( b1[(i+2)/8], (i+2)%8);
	
	        // 1111 1110 is 254
		
		    mask = 254;                                                             // default mask for no bit shift
	
	        mask = (mask >> 8-i%8) | (mask << i%8);                                 // create proper mask to preserve other bits
	
            b2[i/8] = (b2[i/8] & mask) + ( CHECK_BIT(rule, neighboorhood) << i%8);  // use mask and add in new bit
	
	    }
	
	    // RABBIT STEP 2
	
	    for (int i = 0; i < size * 8; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhoodT = CHECK_BIT( b2[(i-2)/8], (i-2)%8) << 3 +    
		         	                       CHECK_BIT( b2[(i-1)/8], (i-1)%8) << 2 +
		    	                           CHECK_BIT( b2[(i+1)/8], (i+1)%8) << 1 +
                                          CHECK_BIT( b2[(i+2)/8], (i+2)%8);
	
	        // 1111 1110 is 254
		
		    mask = 254;                                                             // default mask for no bit shift
	
	        mask = (mask >> 8-i%8) | (mask << i%8);                                 // create proper mask to preserve other bits
	
            b1[i/8] = (b1[i/8] & mask) + ( CHECK_BIT(rule, neighboorhood) << i%8);  // use mask and add in new bit
	
	    }
	
	    // RABBIT STEP 3
	
	    for (int i = 0; i < size * 8; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhoodT = CHECK_BIT( b1[(i-2)/8], (i-2)%8) << 3 +    
		         	                       CHECK_BIT( b1[(i-1)/8], (i-1)%8) << 2 +
		    	                           CHECK_BIT( b1[(i+1)/8], (i+1)%8) << 1 +
                                           CHECK_BIT( b1[(i+2)/8], (i+2)%8);
	
	        // 1111 1110 is 254
		
		    mask = 254;                                                             // default mask for no bit shift
	
	        mask = (mask >> 8-i%8) | (mask << i%8);                                 // create proper mask to preserve other bits
	
            b2[i/8] = (b2[i/8] & mask) + ( CHECK_BIT(rule, neighboorhood) << i%8);  // use mask and add in new bit
	 
	    }
	
	    // RABBIT STEP 4
	
	    for (int i = 0; i < size * 8; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhoodT = CHECK_BIT( b2[(i-2)/8], (i-2)%8) << 3 +    
	    	     	                       CHECK_BIT( b2[(i-1)/8], (i-1)%8) << 2 +
	    		                           CHECK_BIT( b2[(i+1)/8], (i+1)%8) << 1 +
                                           CHECK_BIT( b2[(i+2)/8], (i+2)%8);
	
	        // 1111 1110 is 254
		
		    mask = 254;                                                             // default mask for no bit shift
	
	        mask = (mask >> 8-i%8) | (mask << i%8);                                 // create proper mask to preserve other bits
	
            b1[i/8] = (b1[i/8] & mask) + ( CHECK_BIT(rule, neighboorhood) << i%8);  // use mask and add in new bit
	 
	    }
	
	    // it would be great to detect any sort of cycle here
	
	    // is a1 and b1 the same state just rotated?
		
		// I need a bitshift invariant hash
	
	
	
    }
}

	
	


__global__
void autoStep1(unsigned char *a, unsigned char *b, int *direction, unsigned int ruleBase) {

    unsigned int blockBase = blockIdx.x  * size;

    unsigned int tid = blockBase + threadIdx.x; //(threadIdx.x*8);
	
    // rule per block
    // 256 threads per block so 256 cells in automation 
	
	
	unsigned char neighboorhood = ((a[(threadIdx.x - 2)% size + blockBase]) << 3) +
			                      ((a[(threadIdx.x - 1)% size + blockBase]) << 2) +
	    	                      //((a[(threadIdx.x    )% size + blockBase]) << 2) +
			                      ((a[(threadIdx.x + 1)% size + blockBase]) << 1) +
                                    a[(threadIdx.x + 2)% size + blockBase];
    
	
	b[tid] = CHECK_BIT(blockIdx.x + ruleBase, neighboorhood );
	
    //half one = __float2half(ONE);
	
	if (b[tid] == CHECK_BIT( neighboorhood , 3 )){
			direction[tid] -= 1;}
	if (b[tid] == CHECK_BIT( neighboorhood , 2 )){
			direction[tid] -= 1;}
	if (b[tid] == CHECK_BIT( neighboorhood , 1 )){
			direction[tid] += 1;}
	if (b[tid] == CHECK_BIT( neighboorhood , 0 )){
			direction[tid] += 1;}
			

	/*
	unsigned char ll = a[(threadIdx.x - 2)% 128 + blockBase];
	unsigned char l  = a[(threadIdx.x - 1)% 128 + blockBase];
	unsigned char c  = a[(threadIdx.x    )% 128 + blockBase];
	unsigned char r  = a[(threadIdx.x + 1)% 128 + blockBase];
	unsigned char rr = a[(threadIdx.x + 2)% 128 + blockBase];
  
	b[tid] = (ll << 4) + (l << 3) + (c << 2) + (r << 1) + (rr);//ll * 16 + l * 8 + c * 4 + r * 2 + rr;
 
	b[tid] = CHECK_BIT(blockIdx.x + ruleBase, b[tid]);

		// the original way was b[tid] == l || b[tid] == ll maybe for some weird reason that's better 
	
	
	//if (b[tid] == ll || b[tid] == l){
	//	direction[tid] -= 1;}
	//  if (b[tid] == rr || b[tid] == r){
	//	direction[tid] += 1;}
	
	
	
	if (b[tid] == ll){
			direction[tid] -= 1;}
	if (b[tid] == l){
			direction[tid] -= 1;}
	if (b[tid] == rr){
			direction[tid] += 1;}
	if (b[tid] == r){
			direction[tid] += 1;}
			
    */
			
}

__global__
void autoStep2(unsigned char *a, unsigned char *b, int *direction, unsigned int ruleBase) {

    unsigned int blockBase = blockIdx.x  * size;

    unsigned int tid = blockBase + threadIdx.x;
	
    // rule per block
    // 256 threads per block so 256 cells in automation 
	
	
	unsigned char neighboorhood = ((b[(threadIdx.x - 2)% size + blockBase]) << 3) +
			                      ((b[(threadIdx.x - 1)% size + blockBase]) << 2) +
	    	                      //((b[(threadIdx.x    )% size + blockBase]) << 2) +
			                      ((b[(threadIdx.x + 1)% size + blockBase]) << 1) +
                                   b[(threadIdx.x + 2)% size + blockBase];
    
	a[tid] = CHECK_BIT(blockIdx.x + ruleBase, neighboorhood );
	
    //half one = __float2half(ONE);
	
	if (a[tid] == CHECK_BIT( neighboorhood , 3 )){
			direction[tid] -= 1;}
	if (a[tid] == CHECK_BIT( neighboorhood , 2 )){
			direction[tid] -= 1;}
	if (a[tid] == CHECK_BIT( neighboorhood , 1 )){
			direction[tid] += 1;}
	if (a[tid] == CHECK_BIT( neighboorhood , 0 )){
			direction[tid] += 1;}
			
			
	/*
	
	
	unsigned char ll = b[(threadIdx.x - 2)% 128 + blockBase];
	unsigned char l  = b[(threadIdx.x - 1)% 128 + blockBase];
	unsigned char c  = b[(threadIdx.x    )% 128 + blockBase];
	unsigned char r  = b[(threadIdx.x + 1)% 128 + blockBase];
	unsigned char rr = b[(threadIdx.x + 2)% 128 + blockBase];
  
	a[tid] = (ll << 4) + (l << 3) + (c << 2) + (r << 1) + (rr);//ll * 16 + l * 8 + c * 4 + r * 2 + rr; //ll << 4 + l << 3 + c << 2 + r << 1 + rr;
 
	a[tid] = CHECK_BIT(blockIdx.x + ruleBase, a[tid]);

		// the original way was b[tid] == l || b[tid] == ll maybe for some weird reason that's better
	
	
	//if (a[tid] == ll || a[tid] == l){
	//	direction[tid] -= 1;}
	//if (a[tid] == rr || a[tid] == r){
	//		direction[tid] += 1;}
	
	
	if (a[tid] == ll){
			direction[tid] -= 1;}
	if (a[tid] == l){
			direction[tid] -= 1;}
	if (a[tid] == rr){
			direction[tid] += 1;}
	if (a[tid] == r){
			direction[tid] += 1;}
    */
 
}


__global__ void equals(unsigned char *a, unsigned char *b, unsigned char *resultArray) {

    unsigned int blockBase = blockIdx.x  * size;

    unsigned int tid = blockBase + threadIdx.x;
	
    // rule per block
    // 256 threads per block so 256 cells in automation 
	
	
    if(a[threadIdx.x % size + blockBase] == b[threadIdx.x % size + blockBase]){
        resultArray[threadIdx.x % size + blockBase] = 1;
	}
	else{
        resultArray[threadIdx.x % size + blockBase] = 0;
	}

}

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    hiprand_init(1231+id, id, 0, &state[id]);
}

__global__ void curand_kernel(hiprandState *state,
                                unsigned char *a)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    float x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    x = (hiprand(&localState))%2;
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store last generated result per thread */
    a[id] = (unsigned char) x;
}


//30

// 128  64  32  16   8   4   2   1
//  0   0    0   1   1   1   1   0
 
 
// 1024, 1024, 64
 
int main( int argc, char *argv[] ) {

    // DB allocations

	int rc;
	MDB_env *env;
	MDB_dbi dbi;
	MDB_val key, data;
	MDB_txn *txn;
	MDB_cursor *cursor;
	
	rc = mdb_env_create(&env);
	rc = mdb_env_set_mapsize(env, (68719476736 * 2));
	rc = mdb_env_open(env, argv[2], 0, 0664);


    //
    // Create int arrays on the CPU.
    // ('h' stands for "host".)
    //
    unsigned char ha[N];

    int directionArray[N];
	
    //
    // Create corresponding int arrays on the GPU.
    // ('d' stands for "device".)
    //
    hiprandState *devStates;
    hipMalloc((void **)&devStates, N *  sizeof(hiprandState));
	
    setup_kernel<<<Blocks, size>>>(devStates);
	
	unsigned char *daR, *dbR;
    hipMalloc((void **)&daR, N*sizeof(char));
    hipMalloc((void **)&dbR, N*sizeof(char));
	
	unsigned char *daT, *dbT;
    hipMalloc((void **)&daT, N*sizeof(char));
    hipMalloc((void **)&dbT, N*sizeof(char));

    unsigned int *repeatState;
    hipMalloc((void **)&rpeateState, N*sizeof(char));
	
	
    int *resultArrayD;
    hipMalloc((void **)&resultArrayD, N*sizeof(char));

    //time_point<Clock> start = Clock::now();
	
    //
    // Initialise the input data on the CPU.
    //
	
	//srand(time(NULL));

	int score1[Blocks];
	int score2[Blocks];
    int state[Blocks];
	
	int finalScore[Blocks];
	
    int rule = 0; //atoi( argv[1]);
	//                  2769351843   // 2769351763;  // + 465
	//                  2769352740
	
	// 2769352362
	// 2769352228

	//  

    std::ofstream logs;
	
    int loop = 0;
	
	while (loop < 52 ){
	
		for (int i = 0; i<Blocks; ++i) {
			score1[i] = 0;
			score2[i] = 0;
		}
	
		for (int iters = 0; iters < rounds; iters++) {
	
			srand(time(NULL) + iters);
	
	        // http://stackoverflow.com/questions/14289378/generating-random-numbers-within-cuda-kernel
			
			// GENERATING RANDOM NUMBERS WITH CUDA WILL SPEED THIS UP ALOT MAYBE
	
	
			//for (int i = 0; i<N; ++i) {
			//	ha[i] = rand()%2;
				//if(i%256 - 128 == 0){ ha[i] = 1;}
			//}

			//for (int i = 0; i<N; ++i) {
			//	hb[i] = 0;
			//}
	
			//for (int i = 0; i<N; ++i) {
			//	directionArray[i] = 0;
			//}

			//
			// Copy input data to array on GPU.
			//
			//hipMemcpy(da, ha, N*sizeof(char), hipMemcpyHostToDevice);
			
			hipMemset(da, 0, N * sizeof(char));

            curand_kernel<<<Blocks, size>>>(devStates, da);
			
            //hipMemcpy(ha, da, N*sizeof(char), hipMemcpyDeviceToHost);		
			
			hipMemset(directionArrayD, 0, N*sizeof(int));
			//hipMemcpy(directionArrayD, directionArray, N*sizeof(char), hipMemcpyHostToDevice);
			//
			// Launch GPU code with N threads, one per
			// array element.
			//
	
			int scan = 0;
	
			int sum = 0;
			
			for (int i = 0; i < simDuration; i++) {
			
			    
                // this is the turtle automation, stepping forward one at at time
				autoStep1<<<Blocks, size>>>(daT, dbT, directionArrayD, rule);
				autoStep2<<<Blocks, size>>>(daT, dbT, directionArrayD, rule);
				
				// this is the rabbit automation, stepping forward two at a time
				autoStep1<<<Blocks, size>>>(daR, dbR, directionArrayD, rule);
				autoStep2<<<Blocks, size>>>(daR, dbR, directionArrayD, rule);
				autoStep1<<<Blocks, size>>>(daR, dbR, directionArrayD, rule);
				autoStep2<<<Blocks, size>>>(daR, dbR, directionArrayD, rule);

				// if they produce the same result, then they have found a cycle
				
			}
	
			//hipMemcpy(ha, da, N*sizeof(char), hipMemcpyDeviceToHost);
			hipMemcpy(directionArray, directionArrayD, N*sizeof(int), hipMemcpyDeviceToHost);
	
			sum = 0;
	
			int count = 0;
	
			for (int i = 0+scan; i<N; ++i) {
				if(i%size == 0 && i > 0){
		       
					//score1[count] += sum;
				    //score2[count] += abs(int(sum));
				
				    if (sum > 0){score1[count] = score1[count] + 1;}
					
					if (sum < 0){score2[count] = score2[count] + 1;}
					
					//if (rule + count == 65376){printf("score1: %05d score2: %05d\n", score1[count], score2[count]);}
				
                    //if(sum < 0){printf("auto %ud score2: %d ", rule + count, sum);} //score1[count]);
				
					sum = 0;
					count++;
				}
				
				//if (directionArray[i] < 0 ) {printf("%d", directionArray[i]);}
				sum += directionArray[i];
			}
			
			//count++;
			
			if (sum > 0){score1[count] = score1[count] + 1;}
					
			if (sum < 0){score2[count] = score2[count] + 1;}
			
			//score1[count] += sum;
			//score2[count] += abs(int(sum));
				
		
		}

		
		for (int i = 0; i<(Blocks); i++){
	
			//printf("score1: %8d, score2: %8d", score1[i], score2[i]);

            if (score1[i] > score2[i]){finalScore[i] = score2[i];}
            if (score2[i] > score1[i]){finalScore[i] = score1[i];}
			
			//finalScore[i] = score2[i] - abs(score1[i]); // score1[i] - score2[i];//
	        //if(rule + i == 65376){
			//printf("i:  %d ,rule: %u ,score: %d \n" , i , rule + i, finalScore[i]);}
		
			//if (finalScore[i] > 0) { loop = 0;}
		}
	
        time_point<Clock> end = Clock::now();
	
		//std::cout << "before db" << std::endl;
	
		// write results to db here
		char keyValue[16];
		char dataValue[16];
	
		//std::cout << "db 1" << std::endl;
	
		rc = mdb_txn_begin(env, NULL, 0, &txn);
		rc = mdb_open(txn, NULL, 0, &dbi);
	
		//std::cout << "db 2" << std::endl;
	
		key.mv_size = 8;
		key.mv_data = keyValue;
		data.mv_size = 8;
		data.mv_data = dataValue;
	
		//unsigned int autoID;
	
		//std::cout << "db 3" << std::endl;
	
		for(int i = 0; i < (Blocks); i++){
	
			//std::cout << "1 writing to db #" << i << std::endl;
	
			sprintf(keyValue,   "%08x", rule + i);
			sprintf(dataValue,  "%08x", finalScore[i]);//score1[i]);//
		
			//std::cout << "Key: " << keyValue << " Score: " << dataValue << std::endl;
	
			//std::cout << "2 writing to db #" << i << std::endl;
	
			rc = mdb_put(txn, dbi, &key, &data, 0);
	
		}
		
		rc = mdb_txn_commit(txn);
	
		if (rc) {
			fprintf(stderr, "mdb_txn_commit: (%d) %s\n", rc, mdb_strerror(rc));
			mdb_close(env, dbi);
			mdb_env_close(env);
			return 0;
		}
	
		//printf("MDB commit rc#: %d\n", rc);
	
		//std::cout << "db 4" << std::endl;
	   
	
		if ((rule % 124000) == 0) { 
			printf("Completed section from Rule %d, loop %d\n", rule, loop); 
			logs.open("auto1d_log2.txt");
			logs << "Completed section from Rule %d, loop %d\n", rule, loop;
			logs.close();
		}
		loop++;
	
		rule += Blocks;
	
	}
	
	//for (int i = 0; i<Blocks; i++){
	
	//	printf("auto %u, score: %d\n", rule + i, finalScore[i]);
	//}
	
	//printf("Direction Sum: %d\n", sum);
	
    //milliseconds diff = duration_cast<milliseconds>(end - start);
    //std::cout << diff.count() << "ms" << std::endl;

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}



// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100
// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100