#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "lmdb.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <chrono>
#include <cudaProfiler.h>
#include <lmdb.h>
#include <fstream>
//#include "addressbook.pb.h"


// ./4cell_debug1 65376 123

// vs 

// ./4cell_debug1 0 123

// should be printing out results of same automation but they aint


using Clock = std::chrono::steady_clock;
using std::chrono::time_point;
using std::chrono::duration_cast;
using std::chrono::milliseconds;


//#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include <iostream>
//#include "Utilities.cuh"


//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -llmdb -std=c++11 -o example example.cu
//

#define blockMult 2
#define Blocks (640 * blockMult)
#define size 128
#define N (Blocks * size)
#define CHECK_BIT(var,pos) (var>>pos & 1)
#define time 
#define PLUS_ONE    1.0f
#define MINUS_ONE   -1.0f
#define simDuration 400
#define rounds      100
#define rSize       32

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//

__global__ void print_kernel(unsigned int start, unsigned int *loop, unsigned int ruleBase) {

	
    unsigned int rule = ruleBase +  blockIdx.x * 10 + threadIdx.x; 
	
	if(rule == 9){ rule = 28086;}
	
	//for(int j = 0; j < 5; j++){
	
	//    printf("Hello from block %d, thread %d, rule %d, j %d\n", blockIdx.x, threadIdx.x, rule, j);
	
	//}
	
    unsigned int a1 = start;
    unsigned int a2 = 0;
	
    unsigned int mask;
	
	unsigned char write;
	
	if(rule == 28086){
	    for (int b = 0; b < rSize; b++){
	
	        printf("%d", CHECK_BIT(a1,b));
	
    	}
		
		printf("\n");
	}
	
    for (int j = 0; j < 3; j ++){
	
	    if(rule == 28086){
        printf("1.. %d %d\n", j, rounds);
	    }
	    // TURTLE STEP 1
	
	    for (int i = 0; i < rSize; i++){
		
		    //if(rule == 28086){
		    //printf("2.. %d %d\n", i, rSize);
	        //}
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhood =  (CHECK_BIT( a1, (i-2)%rSize) << 3 )+    
		           	                       (CHECK_BIT( a1, (i-1)%rSize) << 2 )+
		    	                           (CHECK_BIT( a1, (i+1)%rSize) << 1 )+
                                            CHECK_BIT( a1, (i+2)%rSize);
						
						
	        if(rule == 28086){
			     printf("bit1 %d\n", CHECK_BIT( a1, (i-2)%rSize) << 3 );
			     printf("bit2 %d\n", CHECK_BIT( a1, (i-1)%rSize) << 2);
			     printf("bit3 %d\n", CHECK_BIT( a1, (i+1)%rSize) << 1);
			     printf("bit4 %d\n", CHECK_BIT( a1, (i+2)%rSize) );
                 printf("neighboorhood %d\n", neighboorhood);
	        }						
            
	
	        // 1111 1110 is 254
		
	    	mask = 1;                       // defalt mask for no bit shift
	
	        mask = (mask << i);             // create proper mask to preserve other bits
	
        	//if(rule == 28086){
	        //    for (int b = 0; b < rSize; b++){
	
	        //       printf("%d", CHECK_BIT(~mask,b));
	
    	    //}
		
		   //printf("\n");
	       //}
	
            write = CHECK_BIT(rule, neighboorhood);
			
	        if(rule == 28086){
                 printf("%d\n",write);
				 //printf("a2 & mask %d\n", (a2 & mask));
	        }
	
            a2 = (a2 & ~mask);
			
			a2 = a2 + (write << i);
			
        	if(rule == 28086){
	            for (int b = 0; b < rSize; b++){
	
	                printf("%d", CHECK_BIT(a2,b));
	
    	        }
				
				printf("\n");
			
			}

	        //if(rule == 28086){
            //     printf("a2 %d\n", a2);
	        //}
			
			//+ ( write << i);  // use mask and add in new bit
	
            //if(rule == 9){
            //     printf("write %d\n", write);
	        //}
	
	    }
		
		if(rule == 28086){
		     printf("\n");
	    }
	
	    // TURTLE STEP 2
	
 for (int i = 0; i < rSize; i++){
		
		    //if(rule == 28086){
		    //printf("2.. %d %d\n", i, rSize);
	        //}
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhood =  (CHECK_BIT( a2, (i-2)%rSize) << 3 )+    
		           	                       (CHECK_BIT( a2, (i-1)%rSize) << 2 )+
		    	                           (CHECK_BIT( a2, (i+1)%rSize) << 1 )+
                                            CHECK_BIT( a2, (i+2)%rSize);
						
						
	        if(rule == 28086){
			     printf("bit1 %d\n", CHECK_BIT( a2, (i-2)%rSize) << 3 );
			     printf("bit2 %d\n", CHECK_BIT( a2, (i-1)%rSize) << 2);
			     printf("bit3 %d\n", CHECK_BIT( a2, (i+1)%rSize) << 1);
			     printf("bit4 %d\n", CHECK_BIT( a2, (i+2)%rSize) );
                 printf("neighboorhood %d\n", neighboorhood);
	        }						
            
	
	        // 1111 1110 is 254
		
	    	mask = 1;                       // defalt mask for no bit shift
	
	        mask = (mask << i);             // create proper mask to preserve other bits
	
        	//if(rule == 28086){
	        //    for (int b = 0; b < rSize; b++){
	
	        //       printf("%d", CHECK_BIT(~mask,b));
	
    	    //}
		
		   //printf("\n");
	       //}
	
            write = CHECK_BIT(rule, neighboorhood);
			
	        if(rule == 28086){
                 printf("%d\n",write);
				 //printf("a1 & mask %d\n", (a1 & mask));
	        }
	
            a1 = (a1 & ~mask);
			
			a1 = a1 + (write << i);
			
        	if(rule == 28086){
	            for (int b = 0; b < rSize; b++){
	
	                printf("%d", CHECK_BIT(a2,b));
	
    	        }
				
				printf("\n");
			
			}

	        //if(rule == 28086){
            //     printf("a1 %d\n", a1);
	        //}
			
			//+ ( write << i);  // use mask and add in new bit
	
            //if(rule == 9){
            //     printf("write %d\n", write);
	        //}
	
	    }
		
	    if(rule == 28086){
		     printf("\n");
	    }
		
	}
	
	
	
}

__global__ void combinedKernel(unsigned int start, unsigned int *loop, unsigned int ruleBase) {

    if (threadIdx.x == 0) {
        printf("DOES THIS WORK");
    }

 
    unsigned int rule = ruleBase + threadIdx.x; //(threadIdx.x*8);

	
    // rule per block
    // 256 threads per block so 256 cells in automation 
	
	unsigned int a1 = start;
	unsigned int a2;

	unsigned int b1 = start;
	unsigned int b2;
	
	unsigned char write;
	
	unsigned char mask;
	
	    // im going to pack 8 bits in each char to save some memory	
	
        // it's a bit hard to visualize because the bit order is opposite between bits in chars and chars, but I don't think it actually matters
	
	for (int j = 0; j < rounds; j ++){
	
	    // TURTLE STEP 1
	
	    for (int i = 0; i < rSize; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhood = CHECK_BIT( a1, (i-2)%rSize) << 3 +    
		           	                       CHECK_BIT( a1, (i-1)%rSize) << 2 +
		    	                           CHECK_BIT( a1, (i+1)%rSize) << 1 +
                                           CHECK_BIT( a1, (i+2)%rSize);
	
	        // 1111 1110 is 254
		
	    	mask = 254;                                                             // defalt mask for no bit shift
	
	        mask = (mask >> rSize-i) | (mask << i);                                 // create proper mask to preserve other bits
	
            write = CHECK_BIT(rule, neighboorhood);
	
            a2 = (a2 & mask) + ( write << i);  // use mask and add in new bit
	
            if(rule == 9){
			
	            printf("%s",write);
			
			}
	
	    }
		
		printf("\n");
	
	    // TURTLE STEP 2
	
	    for (int i = 0; i < rSize; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhood = CHECK_BIT( a2, (i-2)%rSize) << 3 +    
		        	                       CHECK_BIT( a2, (i-1)%rSize) << 2 +
		    	                           CHECK_BIT( a2, (i+1)%rSize) << 1 +
                                           CHECK_BIT( a2, (i+2)%rSize);
	
	        // 1111 1110 is 254
		
		    mask = 254;                                                             // defalt mask for no bit shift
	
	        mask = (mask >> rSize-i) | (mask << i);                                 // create proper mask to preserve other bits
	
            write = CHECK_BIT(rule, neighboorhood);
	
            a1 = (a1 & mask) + ( write << i);  // use mask and add in new bit
	
            if(rule == 521){
			
	            printf("%s",write);
			
			}
	
	    }
		
		printf("\n");
	
	    // RABBIT STEP 1
	
	    for (int i = 0; i < rSize; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhood =  CHECK_BIT( b1, (i-2)%rSize) << 3 +    
		           	                       CHECK_BIT( b1, (i-1)%rSize) << 2 +
		    	                           CHECK_BIT( b1, (i+1)%rSize) << 1 +
                                           CHECK_BIT( b1, (i+2)%rSize);
	
	        // 1111 1110 is 254
		
	    	mask = 254;                                                             // defalt mask for no bit shift
	
	        mask = (mask >> rSize-i) | (mask << i);                                 // create proper mask to preserve other bits
	
            b2 = (b2 & mask) + ( CHECK_BIT(rule, neighboorhood) << i);  // use mask and add in new bit
	
	    }
	
	    // RABBIT STEP 2
	
	    for (int i = 0; i < rSize; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhood =  CHECK_BIT( b2, (i-2)%rSize) << 3 +    
		           	                       CHECK_BIT( b2, (i-1)%rSize) << 2 +
		    	                           CHECK_BIT( b2, (i+1)%rSize) << 1 +
                                           CHECK_BIT( b2, (i+2)%rSize);
	
	        // 1111 1110 is 254
		
	    	mask = 254;                                                             // defalt mask for no bit shift
	
	        mask = (mask >> rSize-i) | (mask << i);                                 // create proper mask to preserve other bits
	
            b1 = (b1 & mask) + ( CHECK_BIT(rule, neighboorhood) << i);  // use mask and add in new bit
	
	    }
	
	    // RABBIT STEP 3
	
	    for (int i = 0; i < rSize; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhood =  CHECK_BIT( b1, (i-2)%rSize) << 3 +    
		           	                       CHECK_BIT( b1, (i-1)%rSize) << 2 +
		    	                           CHECK_BIT( b1, (i+1)%rSize) << 1 +
                                           CHECK_BIT( b1, (i+2)%rSize);
	
	        // 1111 1110 is 254
		
	    	mask = 254;                                                             // defalt mask for no bit shift
	
	        mask = (mask >> rSize-i) | (mask << i);                                 // create proper mask to preserve other bits
	
            b2 = (b2 & mask) + ( CHECK_BIT(rule, neighboorhood) << i);  // use mask and add in new bit
	
	    }
	
	    // RABBIT STEP 4
	
	    for (int i = 0; i < rSize; i++){
		
	
	        // unpack cells from char and bitshift and add to create neighboorhood number
	    
	        unsigned char neighboorhood =  CHECK_BIT( b2, (i-2)%rSize) << 3 +    
		           	                       CHECK_BIT( b2, (i-1)%rSize) << 2 +
		    	                           CHECK_BIT( b2, (i+1)%rSize) << 1 +
                                           CHECK_BIT( b2, (i+2)%rSize);
	
	        // 1111 1110 is 254
		
	    	mask = 254;                                                             // defalt mask for no bit shift
	
	        mask = (mask >> rSize-i) | (mask << i);                                 // create proper mask to preserve other bits
	
            b1 = (b1 & mask) + ( CHECK_BIT(rule, neighboorhood) << i);  // use mask and add in new bit
	
	    }
		
		// rotate through register and check for cycles
		
		int match = 0;
		
		for (int i =0; i< rSize; i++){
		
		    if(a1 == b1){
			
			    match = 1;
			    break;
			
			}
			
			a1 = (mask >> rSize-1) | (mask << 1);   // rotate to check for symmetries
		
		}
		
		if (match == 1){
		
            printf("Match in %08d, state: %32%d\n", rule , a1);
		    match = a1;
		
		
		
		    // need to run one iteration before checking for match or it will just match right away
		
	        // TURTLE STEP 1
	
	        for (int i = 0; i < rSize; i++){
		
	
	            // unpack cells from char and bitshift and add to create neighboorhood number
	    
	            unsigned char neighboorhood = CHECK_BIT( a1, (i-2)%rSize) << 3 +    
		           	                       CHECK_BIT( a1, (i-1)%rSize) << 2 +
		    	                           CHECK_BIT( a1, (i+1)%rSize) << 1 +
                                           CHECK_BIT( a1, (i+2)%rSize);
	
	            // 1111 1110 is 254
		
	    	    mask = 254;                                                             // defalt mask for no bit shift
	
	            mask = (mask >> rSize-i) | (mask << i);                                 // create proper mask to preserve other bits
	
                a2 = (a2 & mask) + ( CHECK_BIT(rule, neighboorhood) << i);  // use mask and add in new bit
	
	        }
	
	        // TURTLE STEP 2
	
	        for (int i = 0; i < rSize; i++){
		
	
	            // unpack cells from char and bitshift and add to create neighboorhood number
	    
	            unsigned char neighboorhood = CHECK_BIT( a2, (i-2)%rSize) << 3 +    
		        	                       CHECK_BIT( a2, (i-1)%rSize) << 2 +
		    	                           CHECK_BIT( a2, (i+1)%rSize) << 1 +
                                           CHECK_BIT( a2, (i+2)%rSize);
	
	            // 1111 1110 is 254
		
		        mask = 254;                                                             // defalt mask for no bit shift
	
	            mask = (mask >> rSize-i) | (mask << i);                                 // create proper mask to preserve other bits
	
                a1 = (a1 & mask) + ( CHECK_BIT(rule, neighboorhood) << i);  // use mask and add in new bit
	
	        }
		
            loop[rule] = 1;
		
		    while(a1 != match){
		
	            // TURTLE STEP 1
	
	             for (int i = 0; i < rSize; i++){
		
	
	                // unpack cells from char and bitshift and add to create neighboorhood number
	    
	               unsigned char neighboorhood = CHECK_BIT( a1, (i-2)%rSize) << 3 +    
		              	                      CHECK_BIT( a1, (i-1)%rSize) << 2 +
		    	                              CHECK_BIT( a1, (i+1)%rSize) << 1 +
                                              CHECK_BIT( a1, (i+2)%rSize);
	
	                // 1111 1110 is 254
		
 
	                mask = (mask >> rSize-i) | (mask << i);                                 // create proper mask to preserve other bits
	
                    a2 = (a2 & mask) + ( CHECK_BIT(rule, neighboorhood) << i);  // use mask and add in new bit
	
	            }
	
	            // TURTLE STEP 2
	
	            for (int i = 0; i < rSize; i++){
		
	
	                // unpack cells from char and bitshift and add to create neighboorhood number
	    
	                unsigned char neighboorhood = CHECK_BIT( a2, (i-2)%rSize) << 3 +    
		        	                          CHECK_BIT( a2, (i-1)%rSize) << 2 +
		    	                              CHECK_BIT( a2, (i+1)%rSize) << 1 +
                                              CHECK_BIT( a2, (i+2)%rSize);
	
	                // 1111 1110 is 254
		
		            mask = 254;                                                             // defalt mask for no bit shift
	
	                mask = (mask >> rSize-i) | (mask << i);                                 // create proper mask to preserve other bits
	
                    a1 = (a1 & mask) + ( CHECK_BIT(rule, neighboorhood) << i);  // use mask and add in new bit
	
	            }
		
		    loop[rule] = loop[rule] + 1; // increment counter to check for loop duration
		
		    }
		
		}
		
        loop[rule] = a1;
		
	
	    // it would be great to detect any sort of cycle here
	
	    // is a1 and b1 the same state just rotated?
		
		// I need a bitshift invariant hash
	
        // or just put the entire automation in one register
	
	
    }
}



__global__ void equals(unsigned char *a, unsigned char *b, unsigned char *resultArray) {

    unsigned int blockBase = blockIdx.x  * size;

    unsigned int tid = blockBase + threadIdx.x;
	
    // rule per block
    // 256 threads per block so 256 cells in automation 
	
	
    if(a[threadIdx.x % size + blockBase] == b[threadIdx.x % size + blockBase]){
        resultArray[threadIdx.x % size + blockBase] = 1;
	}
	else{
        resultArray[threadIdx.x % size + blockBase] = 0;
	}

}

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    hiprand_init(1231+id, id, 0, &state[id]);
}

__global__ void curand_kernel(hiprandState *state,
                                unsigned char *a)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    float x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    x = (hiprand(&localState))%2;
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store last generated result per thread */
    a[id] = (unsigned char) x;
}


//30

// 128  64  32  16   8   4   2   1
//  0   0    0   1   1   1   1   0
 
 
// 1024, 1024, 64
 
int main( int argc, char *argv[] ) {

    // DB allocations

	int rc;
	MDB_env *env;
	MDB_dbi dbi;
	MDB_val key, data;
	MDB_txn *txn;
	MDB_cursor *cursor;
	
	rc = mdb_env_create(&env);
	rc = mdb_env_set_mapsize(env, (68719476736 * 2));
	rc = mdb_env_open(env, argv[2], 0, 0664);

	
	unsigned int resultArray[Blocks];
	
    unsigned int *resultArrayD;
    hipMalloc((void **)&resultArrayD, Blocks*sizeof(int));

    //time_point<Clock> start = Clock::now();
	
    //
    // Initialise the input data on the CPU.
    //
	
	//srand(time(NULL));

	int score1[Blocks];
	int score2[Blocks];
    int state[Blocks];
	
	int finalScore[Blocks];
	
    int rule = 0; //atoi( argv[1]);
	//                  2769351843   // 2769351763;  // + 465
	//                  2769352740
	
	// 2769352362
	// 2769352228

	//  

    std::ofstream logs;
	
    int loop = 0;
	
	unsigned int randomStart = 0;

    for (int i = 0; i<Blocks; ++i) {
			score1[i] = 0;
			score2[i] = 0;
			finalScore[i] = 0;
    }
	
	while (loop < 2 ){
	
       
	
		for (int iters = 0; iters < 2; iters++) {
	
			srand(time(0) + iters);
	
            randomStart = rand();
			
			hipMemset(resultArrayD, 0, Blocks*sizeof(int));

			// Launch GPU code with N threads, one per
			// array element.

	
			int scan = 0;
	
			int sum = 0;
			
			//printf("~~~~~~~~~~~ start: %d\n", randomStart);
            //printf("~~~~~~~~~~~ rule : %d\n", rule);
			
			//combinedKernel<<<Blocks, size>>>(randomStart, resultArrayD, rule);
	
            //hipDeviceSynchronize();
			
            print_kernel<<<10, 10>>>(randomStart, resultArrayD, rule);
            hipDeviceSynchronize();
	
			hipMemcpy(resultArray, resultArrayD, Blocks*sizeof(int), hipMemcpyDeviceToHost);
	
	
	
			sum = 0;
	
			int count = 0;
	
			for (int i = 0; i<Blocks; ++i) {

                //printf("Result for %08d: %d\n", i + rule, resultArray[i]);
                score1[i] += resultArray[i] / rounds;
				
				//if (directionArray[i] < 0 ) {printf("%d", directionArray[i]);}
			}
			
			//count++;
			
			if (sum > 0){score1[count] = score1[count] + 1;}
					
			if (sum < 0){score2[count] = score2[count] + 1;}
			
			//score1[count] += sum;
			//score2[count] += abs(int(sum));
				
		
		}

        printf("probe1\n");
	
 
	
        time_point<Clock> end = Clock::now();
	
		//std::cout << "before db" << std::endl;
	
		// write results to db here
		char keyValue[16];
		char dataValue[16];
	
		//std::cout << "db 1" << std::endl;
	
		rc = mdb_txn_begin(env, NULL, 0, &txn);
		rc = mdb_open(txn, NULL, 0, &dbi);
	
		//std::cout << "db 2" << std::endl;
	
		key.mv_size = 8;
		key.mv_data = keyValue;
		data.mv_size = 8;
		data.mv_data = dataValue;
	
		//unsigned int autoID;
	
		//std::cout << "db 3" << std::endl;
	
		for(int i = 0; i < (Blocks); i++){
	
			//std::cout << "1 writing to db #" << i << std::endl;
	
			sprintf(keyValue,   "%08x", rule + i);
			sprintf(dataValue,  "%08x", score1[i]);//
		
			//std::cout << "Key: " << keyValue << " Score: " << dataValue << std::endl;
	
			//std::cout << "2 writing to db #" << i << std::endl;
	
			rc = mdb_put(txn, dbi, &key, &data, 0);
	
		}
		
		rc = mdb_txn_commit(txn);
	
		if (rc) {
			fprintf(stderr, "mdb_txn_commit: (%d) %s\n", rc, mdb_strerror(rc));
			mdb_close(env, dbi);
			mdb_env_close(env);
			return 0;
		}
	
		//printf("MDB commit rc#: %d\n", rc);
	
		//std::cout << "db 4" << std::endl;
	   
	
		if ((rule % 124000) == 0) { 
			printf("Completed section from Rule %d, loop %d\n", rule, loop); 
			logs.open("auto1d_log2.txt");
			logs << "Completed section from Rule %d, loop %d\n", rule, loop;
			logs.close();
		}
		loop++;
	
		rule += Blocks;
	
	}
	
	//for (int i = 0; i<Blocks; i++){
	
	//	printf("auto %u, score: %d\n", rule + i, finalScore[i]);
	//}
	
	//printf("Direction Sum: %d\n", sum);
	
    //milliseconds diff = duration_cast<milliseconds>(end - start);
    //std::cout << diff.count() << "ms" << std::endl;

    //
    // Free up the arrays on the GPU.
    //
    hipFree(resultArrayD);

    return 0;
}



// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100
// 0011001100100011001100110010001000110011001000110011001100100100100100100100100100100100100100100100